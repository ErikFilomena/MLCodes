#include "hip/hip_runtime.h"
#include "LinearAlgebra.h"

#include "hip/hip_runtime.h"
#include ""
#include "CudaStuff.h"
//#include "Matrix.h"
#include <stdio.h>


__global__ // Multiplies A*B, where A nAxm is row major and B mxmB is col-major dst row-major nAxmB
void _cudaMatrixMult(NUMBER* dst, NUMBER* A, NUMBER* B, size_t nA,size_t m,size_t mB){
    size_t rowOffset = (blockIdx.x*blockDim.x +threadIdx.x);
	size_t colOffset = (blockIdx.y*blockDim.y +threadIdx.y);
	if(rowOffset < nA && colOffset < mB){
		size_t offsetA = rowOffset*m;
		size_t offsetB = colOffset*m;
		size_t offsetDst = rowOffset*mB + colOffset;
		dst[offsetDst]=0;
		for(size_t i=0;i< m;i++){
			dst[offsetDst] += A[offsetA + i]*B[offsetB+i];
		}
	}
	
}

int cudaMatrixMult(NUMBER* dst, NUMBER* A, NUMBER* B, size_t nA,size_t m,size_t mB){
	
	int nXBlocks = nA/32 + ((nA%32)!=0);
	int nYBlocks = mB/32 + ((mB%32)!=0);

	dim3 blocks(nXBlocks,nYBlocks);
	dim3 threads(32,32);

	_cudaMatrixMult<<<blocks,threads>>>(dst,A,B,nA,m,mB);
	return 0;
}