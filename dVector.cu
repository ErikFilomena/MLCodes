#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "VectorCuda.h"
#include "CudaStuff.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>



//DEVICE FUNCTIONS
__global__
KERNEL _dVectorAdd(double* dst, double* src1, double* src2, size_t len){
    size_t index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index<len){
        dst[index] = src1[index]+src2[index];
    }
}

__global__
KERNEL _dVectorSub(double* dst, double* src1, double* src2, size_t len){
    size_t index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index<len){
        dst[index] = src1[index]-src2[index];
    }
}

__global__
KERNEL _dVectorHadamard(double* dst, double* src1, double* src2, size_t len){
    size_t index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index<len){
        dst[index] = src1[index]*src2[index];
    }
}

__global__
KERNEL _dVectorDot(double* result, double* src1, double* src2, size_t len){
    size_t index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index<len){
        double val = src1[index]*src2[index];
        atomicAdd(result,val);
    }
}

__global__
KERNEL _dVectorAssign(double* dst, double val, size_t len){
    size_t index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index<len){
        dst[index] = val;
    }
}




//HOST WRAPPERS
void CudaStuff::dVectorAdd(double* dst, double* src1, double* src2, size_t len){
    int nBlocks = len/MAX_THREADS + 1; 
     _dVectorAdd<<<nBlocks,MAX_THREADS>>>(dst,src1,src2,len);
}

void CudaStuff::dVectorSub(double* dst, double* src1, double* src2, size_t len){
    int nBlocks = len/MAX_THREADS + 1; 
     _dVectorSub<<<nBlocks,MAX_THREADS>>>(dst,src1,src2,len);
}


void CudaStuff::dVectorHadamard(double* dst, double* src1, double* src2, size_t len){
    int nBlocks = len/MAX_THREADS + 1; 
     _dVectorHadamard<<<nBlocks,MAX_THREADS>>>(dst,src1,src2,len);
}

double CudaStuff::dVectorDot(double* src1, double* src2, size_t len){
    int nBlocks = len/MAX_THREADS + 1; 
    double* ptrResult;
    double result;
    hipMalloc(&ptrResult,sizeof(double));
    _dVectorDot<<<nBlocks,MAX_THREADS>>>(ptrResult,src1,src2,len);
    hipMemcpy(&result,ptrResult,sizeof(double),hipMemcpyDeviceToHost);
    hipFree(ptrResult);
    return result;
}

void* CudaStuff::ToCuda(void* data, size_t len, size_t byteSize){
	void* temp;
	size_t nBytes =  len*byteSize ;
	hipMalloc( &temp, nBytes );
	hipMemcpy(temp,data,nBytes,hipMemcpyHostToDevice);
	return temp;
}

void CudaStuff::ToHost(void* data, size_t len, size_t byteSize, void* dst){
	hipMemcpy(dst,data,len*byteSize,hipMemcpyDeviceToHost);
}

void CudaStuff::UnCuda(void* data){
	hipFree(data);
}

double* CudaStuff::CudaInit(size_t n){
    double* temp;
    hipMallocManaged(&temp, n*sizeof(double));
    return temp;
}

void CudaStuff::dVectorAssign(double* dst, double val, size_t len){
    size_t nBlocks = len/MAX_THREADS + 1; 
    _dVectorAssign<<<nBlocks,MAX_THREADS>>>(dst,val,len);
}
