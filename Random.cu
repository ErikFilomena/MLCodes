
#include <hip/hip_runtime.h>
/*#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "VectorCuda.h"
#include "CudaStuff.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cuda.h>
#include <curand.h>

#include "RandomCuda.h"

__global__ 
KERNEL _MonteCarloTest(double* result,size_t len){
	size_t index = threadIdx.x + blockIdx.x*blockDim.x;
	if(index < len){
		result[index] = exp(result[index]) + 1;
	}
}

void dGenerateRandomVector(double** target, size_t len){
	double* temp;
	cudaMalloc(&temp,len*sizeof(double));

	curandGenerator_t generator;
	curandCreateGenerator(&generator, CURAND_RNG_PSEUDO_DEFAULT);
	curandSetPseudoRandomGeneratorSeed(generator,time(0));
	curandGenerateUniformDouble(generator,temp, len);
	curandDestroyGenerator(generator);

	size_t nBlocks = len/MAX_THREADS + 1;

	_MonteCarloTest<<<nBlocks,MAX_THREADS>>>(temp,len);
	cudaDeviceSynchronize();


	cudaMemcpy(*target, temp, len*sizeof(double),cudaMemcpyDeviceToHost);

	cudaFree(temp);
	
}*/